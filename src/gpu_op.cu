#include "hip/hip_runtime.h"
#include "./c_runtime_api.h"
#include <cassert>
#include <cstdio>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <>
#include <algorithm>
//��
#include <hipDNN.h>
#include <stdlib.h>
#include <iostream>
//��-
using namespace std;
#define MAX_THREADS_NUM 512
#define MAX_BLOCKS_NUM 4096
#define BLOCK_NUM(count) min(((count + MAX_THREADS_NUM - 1) / MAX_THREADS_NUM), MAX_BLOCKS_NUM)
#define CUDA_1D_KERNEL_LOOP(i, n) \
  for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < (n); \
        i += blockDim.x * gridDim.x)

//��
#define CUDNN_CALL(f) { \
hipdnnStatus_t err = (f); \
if (err != HIPDNN_STATUS_SUCCESS) {\
    \
        std::cout << "    Error occurred: " << err << std::endl; \
        std::exit(1); \
} \
}
//��-
__global__ void matrix_array_set_kernel(int count,
                                        float *arr,
                                        float value) {
  CUDA_1D_KERNEL_LOOP(index, count) {
    arr[index] = value;
  }
}

__global__ void matrix_broadcast_to_kernel(int inputCount, float* inputArr,
                                           int outputCount, float* outputArr) {
  CUDA_1D_KERNEL_LOOP(index, outputCount) {
      outputArr[index] = inputArr[index % inputCount];
  }
}

__global__ void matrix_reduce_sum_axis_zero_kernel(float* inputArr,
                                                   int outputCount, float* outputArr,
                                                   int zeroDim) {
      CUDA_1D_KERNEL_LOOP(index, outputCount) {
          float sum = 0;
          for (int i = 0; i < zeroDim; ++i) {
              sum += inputArr[index + i * outputCount];
          }
          outputArr[index] = sum;
      }
}

__global__ void matrix_elementwise_add_kernel(float* matAData, float* matBData,
                                              float* outputData, int count) {
    CUDA_1D_KERNEL_LOOP(index, count) {
        outputData[index] = matAData[index] + matBData[index];
    }
}

__global__ void matrix_elementwise_add_by_const_kernel(float* inputArr, float val,
                                                       float* outputArr, int count) {
    CUDA_1D_KERNEL_LOOP(index, count) {
        outputArr[index] = inputArr[index] + val;
    }
}

__global__ void matrix_elementwise_multiply_kernel(float* matAData, float* matBData,
                                                   float* outputData, int count) {
    CUDA_1D_KERNEL_LOOP(index, count) {
        outputData[index] = matAData[index] * matBData[index];
    }
}

__global__ void matrix_elementwise_multipy_by_const_kernel(float* inputArr, float val,
                                                           float* outputArr, int count) {
    CUDA_1D_KERNEL_LOOP(index, count) {
        outputArr[index] = inputArr[index] * val;
    }
}

__global__ void matrix_relu_kernel(float* inputArr, float* outputArr, int count) {
    CUDA_1D_KERNEL_LOOP(index, count) {
        outputArr[index] = inputArr[index];
        if (inputArr[index] < 0) {
            outputArr[index] = 0.f;
        }
    }
}

__global__ void matrix_relu_gradient_kernel(const float* inputArr, const float* gradArr,
                                            float* outputArr, int count) {
    CUDA_1D_KERNEL_LOOP(index, count) {
        outputArr[index] = inputArr[index] > 0 ? gradArr[index] : 0;
    }
}

__global__ void matrix_softmax_kernel(int nRow, int nCol, float* inputArr, float* outputArr) {
    int y = blockIdx.x * blockDim.x + threadIdx.x;
    if (y >= nRow) return;

    float* input = inputArr + y * nCol;
    float* output = outputArr + y * nCol;

    float maxval = *input;
    for (int i = 1; i < nCol; ++i) {
        maxval = max(input[i], maxval);
    }
    float sum = 0;
    for (int i = 0; i < nCol; ++i) {
        sum += expf(input[i] - maxval);
    }
    for (int i = 0; i < nCol; ++i) {
        output[i] = expf(input[i] - maxval) / sum;
    }
}

/* all your GPU kernel code, e.g. matrix_softmax_cross_entropy_kernel */

// y = inputs[0], y_ = inputs[1]
// np.mean(-np.sum(y_ * np.log(softmax(y)), axis=1), keepdims=True)
__global__ void matrix_softmax_cross_entropy_kernel(int nrow, int ncol,
                                                    const float *input_a,
                                                    const float *input_b,
                                                    float *output) {
  // Dynamic shared memory, size provided at kernel launch.
  extern __shared__ float loss_per_row[];
  // Two dimensional thread blocks.
  int y = blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x +
          threadIdx.x;
  if (y >= nrow) {
    return;
  }
  input_a += y * ncol;
  input_b += y * ncol;
  float maxval = *input_a;
  // Find max for a row.
  for (int x = 1; x < ncol; ++x) {
    maxval = max(maxval, input_a[x]);
  }
  // Deduct by max for a row, and raise to exp.
  float sum = 0;
  for (int x = 0; x < ncol; ++x) {
    sum += exp(input_a[x] - maxval);
  }
  // Compute per-row loss.
  float loss = 0;
  for (int x = 0; x < ncol; ++x) {
    loss -= input_b[x] * log(exp(input_a[x] - maxval) / sum);
  }
  loss_per_row[y] = loss;
  __syncthreads();
  // Compute reduce_mean across rows.
  float mean_loss = 0;
  // Use a single thread to reduce mean across rows.
  if ((threadIdx.x == 0) && (threadIdx.y == 0)) {
    for (int i = 0; i < nrow; ++i) {
      mean_loss += loss_per_row[i];
    }
    mean_loss /= nrow;
    output[0] = mean_loss;
  }
}

int DLGpuArraySet(DLArrayHandle arr, float value) {
  int count = 1;
  for (int i = 0; i < arr->ndim; ++i) {
    count *= arr->shape[i];
  }
  float *arr_data = (float *)arr->data;
  matrix_array_set_kernel<<<BLOCK_NUM(count), MAX_THREADS_NUM>>>(
    count, arr_data, value);
  return 0;
}

int DLGpuBroadcastTo(const DLArrayHandle input, DLArrayHandle output) {
  assert(input->ndim + 1 == output->ndim);
  int inputCount = 1, outputCount = output->shape[0];
  for (int i = 0; i < input->ndim; ++i) {
      assert(input->shape[i] == output->shape[i + 1]);
      inputCount *= input->shape[i];
      outputCount *= output->shape[i + 1];
  }
  float* inputArr = (float*) input->data;
  float* outputArr = (float*) output->data;
  matrix_broadcast_to_kernel<<<BLOCK_NUM(outputCount), MAX_THREADS_NUM>>>(
    inputCount, inputArr, outputCount, outputArr);
  return 0;
}

int DLGpuReduceSumAxisZero(const DLArrayHandle input, DLArrayHandle output) {
  assert(input->ndim == output->ndim + 1);
  int zeroDim = input->shape[0], outputCount = 1;
    for (int i = 0; i < output->ndim; ++i) {
        assert(input->shape[i+1] == output->shape[i]);
        outputCount *= output->shape[i];
    }
  float* inputArr = (float*) input->data;
  float* outputArr = (float*) output->data;
  matrix_reduce_sum_axis_zero_kernel<<<BLOCK_NUM(outputCount), MAX_THREADS_NUM>>>(
          inputArr, outputCount, outputArr, zeroDim);
  return 0;
}

int DLGpuMatrixElementwiseAdd(const DLArrayHandle matA,
                              const DLArrayHandle matB, DLArrayHandle output) {
  assert(matA->ndim == output->ndim);
  assert(matB->ndim == output->ndim);
  int count = 1;
  for (int i = 0; i < matA->ndim; ++i) {
    assert(matA->shape[i] == output->shape[i]);
    assert(matB->shape[i] == output->shape[i]);
    count *= matA->shape[i];
  }
  float* matAData = (float*) matA->data;
  float* matBData = (float*) matB->data;
  float* outputData = (float*) output->data;
  matrix_elementwise_add_kernel<<<BLOCK_NUM(count), MAX_THREADS_NUM>>>(
          matAData, matBData, outputData, count);
  return 0;
}

int DLGpuMatrixElementwiseAddByConst(const DLArrayHandle input, float val,
                                     DLArrayHandle output) {
  assert(input->ndim == output->ndim);
  int count = 1;
  for (int i = 0; i < input->ndim; ++i) {
    assert(input->shape[i] == output->shape[i]);
    count *= input->shape[i];
  }
  float* inputArr = (float*) input->data;
  float* outputArr = (float*) output->data;
  matrix_elementwise_add_by_const_kernel<<<BLOCK_NUM(count), MAX_THREADS_NUM>>>(
          inputArr, val, outputArr, count);
  return 0;
}

int DLGpuMatrixElementwiseMultiply(const DLArrayHandle matA,
                                   const DLArrayHandle matB,
                                   DLArrayHandle output) {
  assert(matA->ndim == output->ndim);
  assert(matB->ndim == output->ndim);
  int count = 1;
  for (int i = 0; i < matA->ndim; ++i) {
    assert(matA->shape[i] == output->shape[i]);
    assert(matB->shape[i] == output->shape[i]);
    count *= matA->shape[i];
  }
  float* matAData = (float*) matA->data;
  float* matBData = (float*) matB->data;
  float* outputData = (float*) output->data;
  matrix_elementwise_multiply_kernel<<<BLOCK_NUM(count), MAX_THREADS_NUM>>>(
          matAData, matBData, outputData, count);
  return 0;
}

int DLGpuMatrixMultiplyByConst(const DLArrayHandle input, float val,
                               DLArrayHandle output) {
  assert(input->ndim == output->ndim);
  int count = 1;
  for (int i = 0; i < input->ndim; ++i) {
    assert(input->shape[i] == output->shape[i]);
    count *= input->shape[i];
  }
  float* inputArr = (float*) input->data;
  float* outputArr = (float*) output->data;
  matrix_elementwise_multipy_by_const_kernel<<<BLOCK_NUM(count), MAX_THREADS_NUM>>>(
          inputArr, val, outputArr, count);
  return 0;
}

int DLGpuMatrixMultiply(const DLArrayHandle matA, bool transposeA,
                        const DLArrayHandle matB, bool transposeB,
                        DLArrayHandle matC) {
  // Hint: use cublas
  // cublas assume matrix is column major
  assert(matA->ndim == 2);
  assert(matB->ndim == 2);
  assert(matC->ndim == 2);
  assert(matA->shape[transposeA ? 0 : 1] == matB->shape[transposeB ? 1 : 0]);
  assert(matA->shape[transposeA ? 1 : 0] == matC->shape[0]);
  assert(matB->shape[transposeB ? 0 : 1] == matC->shape[1]);

  hipblasHandle_t handle;
  hipblasCreate(&handle);
  const float* matAData = (const float*) matA->data;
  const float* matBData = (const float*) matB->data;
  float* matCData = (float*) matC->data;
  float alpha = 1, beta = 0;

  hipblasSgemm(handle,
              (transposeB ? HIPBLAS_OP_T : HIPBLAS_OP_N),
              (transposeA ? HIPBLAS_OP_T : HIPBLAS_OP_N),
              (transposeB ? matB->shape[0] : matB->shape[1]),
              (transposeA ? matA->shape[1] : matA->shape[0]),
              (transposeB ? matB->shape[1] : matB->shape[0]),
              &alpha,
              matBData, matB->shape[1],
matAData, matA->shape[1],
& beta,
matCData, (transposeB ? matB->shape[0] : matB->shape[1]));

return 0;
}

int DLGpuRelu(const DLArrayHandle input, DLArrayHandle output) {
    assert(input->ndim == output->ndim);
    int count = 1;
    for (int i = 0; i < input->ndim; ++i) {
        assert(input->shape[i] == output->shape[i]);
        count *= input->shape[i];
    }
    float* inputArr = (float*)input->data;
    float* outputArr = (float*)output->data;
    matrix_relu_kernel << <BLOCK_NUM(count), MAX_THREADS_NUM >> > (
        inputArr, outputArr, count);
    return 0;
}

int DLGpuReluGradient(const DLArrayHandle input, const DLArrayHandle in_grad,
    DLArrayHandle output) {
    assert(input->ndim == in_grad->ndim);
    assert(input->ndim == output->ndim);
    int count = 1;
    for (int i = 0; i < input->ndim; ++i) {
        assert(input->shape[i] == in_grad->shape[i]);
        assert(input->shape[i] == output->shape[i]);
        count *= input->shape[i];
    }
    const float* inputArr = (const float*)input->data;
    const float* gradArr = (const float*)in_grad->data;
    float* outputArr = (float*)output->data;
    matrix_relu_gradient_kernel << <BLOCK_NUM(count), MAX_THREADS_NUM >> > (
        inputArr, gradArr, outputArr, count);
    return 0;
}

int DLGpuSoftmax(const DLArrayHandle input, DLArrayHandle output) {
    assert(input->ndim == 2);
    assert(output->ndim == 2);
    assert(input->shape[0] == output->shape[0]);
    assert(input->shape[1] == output->shape[1]);

    int nRow = input->shape[0];
    int nCol = input->shape[1];

    dim3 block(MAX_THREADS_NUM);
    dim3 grid((nRow + block.x - 1) / block.x);

    float* inputArr = (float*)input->data;
    float* outputArr = (float*)output->data;

    matrix_softmax_kernel << <grid, block >> > (nRow, nCol, inputArr, outputArr);

    return 0;
}

int DLGpuSoftmaxCrossEntropy(const DLArrayHandle input_a,
    const DLArrayHandle input_b,
    DLArrayHandle output) {
    assert(input_a->ndim == 2);
    assert(input_b->ndim == 2);
    assert(output->ndim == 1);
    assert(input_a->shape[0] == input_b->shape[0] &&
        input_a->shape[1] == input_b->shape[1]);
    int nrow = input_a->shape[0];
    // Maximum x- or y-dimension of a block = 1024
    // But we need 'nrow' shared memory, and max shared memory is 48KB.
    // Conservatively allow max 16KB shared memory.
    assert(nrow <= 1024 * 4);
    int ncol = input_a->shape[1];
    const float* input_data_a = (const float*)input_a->data;
    const float* input_data_b = (const float*)input_b->data;
    float* output_data = (float*)output->data;
    dim3 threads;
    if (nrow <= 1024) {
        threads.x = nrow;
    }
    else {
        threads.x = 1024;
        threads.y = (nrow + 1023) / 1024;
    }
    // 1 block, each block with 'threads' number of threads with 'nrow' shared
    // memory size
    matrix_softmax_cross_entropy_kernel << <1, threads, nrow * sizeof(float) >> > (
        nrow, ncol, input_data_a, input_data_b, output_data);
    return 0;
}




//3ά
int DLGpuConvolution1DForward(const DLArrayHandle input,
    const DLArrayHandle filter,
    DLArrayHandle output,
    hipdnnTensorFormat_t dataformat,
    const paddingStatus_t padding,
    const int v          /*filter stride */) {

    cout<<dataformat<<endl;
    cout<<padding<<endl;
    assert(input->ndim == 3);
    assert(filter->ndim == 3);



    int input_n = input->shape[0];
    int input_c = input->shape[2];
    int input_h = 1;
    int input_w = input->shape[1];

    int filter_n = filter->shape[0];
    int filter_c = filter->shape[2];
    int filter_h = 1;
    int filter_w = filter->shape[1];

    if (dataformat == 0) {
        input_n = input->shape[0];
        input_c = input->shape[1];
        input_h = 1;
        input_w = input->shape[2];

        filter_n = filter->shape[0];
        filter_c = filter->shape[1];
        filter_h = 1;
        filter_w = filter->shape[2];
    }

    int out_n;
    int out_c;
    int out_h;
    int out_w;

    int pad_h = 0;
    int pad_w = 0;

    int u = 1;

    if (padding == 1) {
        pad_w = filter_w / 2;
    }


    //handle
    hipdnnHandle_t handle;
    CUDNN_CALL(hipdnnCreate(&handle));
     printf("0\n");
    //input
    hipdnnTensorDescriptor_t input_descriptor;
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&input_descriptor));
    CUDNN_CALL(hipdnnSetTensor4dDescriptor(input_descriptor,
        dataformat,
        HIPDNN_DATA_FLOAT,
        input_n,
        input_c,
        input_h,
        input_w));


    //������
    hipdnnFilterDescriptor_t filter_descriptor;
    CUDNN_CALL(hipdnnCreateFilterDescriptor(&filter_descriptor));
    CUDNN_CALL(hipdnnSetFilter4dDescriptor(filter_descriptor,
        HIPDNN_DATA_FLOAT,
        dataformat,
        filter_n,
        filter_c,
        filter_h,
        filter_w));


    //��ķ�ʽ��������padding
    hipdnnConvolutionDescriptor_t conv_descriptor;
    CUDNN_CALL(hipdnnCreateConvolutionDescriptor(&conv_descriptor));
    CUDNN_CALL(hipdnnSetConvolution2dDescriptor(conv_descriptor,
        pad_h, pad_w, // zero-padding
        u, v, // stride
        1, 1,
        HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));

    //����output��4��ά��
    CUDNN_CALL(hipdnnGetConvolution2dForwardOutputDim(
        conv_descriptor,
        input_descriptor,
        filter_descriptor,
        &out_n,
        &out_c,
        &out_h,
        &out_w));

    if (dataformat == 0) {
        assert(output->shape[0] == out_n);
        assert(output->shape[1] == out_c);
        assert(1 == out_h);
        assert(output->shape[2] == out_w);
    }
    else {
        assert(output->shape[0] == out_n);
        assert(output->shape[2] == out_c);
        assert(1 == out_h);
        assert(output->shape[1] == out_w);
    }



    //output��Ϣ
    hipdnnTensorDescriptor_t output_descriptor;
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&output_descriptor));
    CUDNN_CALL(hipdnnSetTensor4dDescriptor(output_descriptor,
        dataformat,
        HIPDNN_DATA_FLOAT,
        out_n,
        out_c,
        out_h,
        out_w));

    //�������㷨
    hipdnnConvolutionFwdAlgo_t algo;
    CUDNN_CALL(hipdnnGetConvolutionForwardAlgorithm(handle,
        input_descriptor,
        filter_descriptor,
        conv_descriptor,
        output_descriptor,
        HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST,
        0,
        &algo));

    //׼����������Ŀռ�
    size_t workspace_size = 0;
    CUDNN_CALL(hipdnnGetConvolutionForwardWorkspaceSize(handle,
        input_descriptor,
        filter_descriptor,
        conv_descriptor,
        output_descriptor,
        algo,
        &workspace_size));
    void* workspace = nullptr;
    hipMalloc(&workspace, workspace_size);

    printf("6\n");
    // convolution
    auto alpha = 1.0f, beta = 0.0f;
    CUDNN_CALL(hipdnnConvolutionForward(handle,
        &alpha, //x*w����
        input_descriptor,
        input->data,
        filter_descriptor,
        filter->data,
        conv_descriptor,
        algo,
        workspace,
        workspace_size,
        &beta, //y����,y�������ݽ������ţ�
        output_descriptor,
        output->data));
    //�ڴ�
    hipFree(workspace);
    CUDNN_CALL(hipdnnDestroyTensorDescriptor(input_descriptor));
    CUDNN_CALL(hipdnnDestroyTensorDescriptor(output_descriptor));
    CUDNN_CALL(hipdnnDestroyConvolutionDescriptor(conv_descriptor));
    CUDNN_CALL(hipdnnDestroyFilterDescriptor(filter_descriptor));
    CUDNN_CALL(hipdnnDestroy(handle));

    return 0;

}


int DLGpuConvolution1DForwardGetOutShape(const int* input_shapes,
    const int* filter_shapes,
    int* output_shapes,
    hipdnnTensorFormat_t dataformat,
    const paddingStatus_t padding,
    const int v          /*filter stride */) {

    cout << dataformat << endl;
    cout << padding << endl;


    int input_n = input_shapes[0];
    int input_c = input_shapes[2];
    int input_h = 1;
    int input_w = input_shapes[1];

    int filter_n = filter_shapes[0];
    int filter_c = filter_shapes[2];
    int filter_h = 1;
    int filter_w = filter_shapes[1];

    if (dataformat == 0) {
        input_n = input_shapes[0];
        input_c = input_shapes[1];
        input_h = 1;
        input_w = input_shapes[2];

        filter_n = filter_shapes[0];
        filter_c = filter_shapes[1];
        filter_h = 1;
        filter_w = filter_shapes[2];
    }



    int out_n;
    int out_c;
    int out_h;
    int out_w;

    int pad_h = 0;
    int pad_w = 0;

    int u = 1;

    if (padding == 1) {
        pad_w = filter_w / 2;
    }


    //handle
    hipdnnHandle_t handle;
    CUDNN_CALL(hipdnnCreate(&handle));

    //input
    hipdnnTensorDescriptor_t input_descriptor;
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&input_descriptor));
    CUDNN_CALL(hipdnnSetTensor4dDescriptor(input_descriptor,
        dataformat,
        HIPDNN_DATA_FLOAT,
        input_n,
        input_c,
        input_h,
        input_w));


    //������
    hipdnnFilterDescriptor_t filter_descriptor;
    CUDNN_CALL(hipdnnCreateFilterDescriptor(&filter_descriptor));
    CUDNN_CALL(hipdnnSetFilter4dDescriptor(filter_descriptor,
        HIPDNN_DATA_FLOAT,
        dataformat,
        filter_n,
        filter_c,
        filter_h,
        filter_w));


    //��ķ�ʽ��������padding
    hipdnnConvolutionDescriptor_t conv_descriptor;
    CUDNN_CALL(hipdnnCreateConvolutionDescriptor(&conv_descriptor));
    CUDNN_CALL(hipdnnSetConvolution2dDescriptor(conv_descriptor,
        pad_h, pad_w, // zero-padding
        u, v, // stride
        1, 1,
        HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));

    //����output��4��ά��
    CUDNN_CALL(hipdnnGetConvolution2dForwardOutputDim(
        conv_descriptor,
        input_descriptor,
        filter_descriptor,
        &out_n,
        &out_c,
        &out_h,
        &out_w));



    if (dataformat == 0) {
        output_shapes[0] = out_n;
        output_shapes[1] = out_c;
        output_shapes[2] = out_w;
    }
    else {
        output_shapes[0] = out_n;
        output_shapes[1] = out_w;
        output_shapes[2] = out_c;
    }



    CUDNN_CALL(hipdnnDestroyTensorDescriptor(input_descriptor));
    CUDNN_CALL(hipdnnDestroyConvolutionDescriptor(conv_descriptor));
    CUDNN_CALL(hipdnnDestroyFilterDescriptor(filter_descriptor));
    CUDNN_CALL(hipdnnDestroy(handle));

    return 0;

}


int DLGpuConvolution1DBackward(const DLArrayHandle input,
    const DLArrayHandle doutput,
    const DLArrayHandle filter,
    DLArrayHandle dfilter,
    DLArrayHandle dinput,
    hipdnnTensorFormat_t dataformat,
    const paddingStatus_t padding,
    const int v          /*filter stride */) {

    assert(input->ndim == 3);
    assert(filter->ndim == 3);


     int input_n = input->shape[0];
     int input_c = input->shape[2];
     int input_h = 1;
     int input_w = input->shape[1];

     int filter_n = filter->shape[0];
     int filter_c = filter->shape[2];
     int filter_h = 1;
     int filter_w = filter->shape[1];


    if (dataformat == 0) {
        input_n = input->shape[0];
        input_c = input->shape[1];
        input_h = 1;
        input_w = input->shape[2];

        filter_n = filter->shape[0];
        filter_c = filter->shape[1];
        filter_h = 1;
        filter_w = filter->shape[2];
    }

  

    int out_n;
    int out_c;
    int out_h;
    int out_w;

    int pad_h = 0;
    int pad_w = 0;

    int u = 1;

    if (padding == 1) {
        pad_w = filter_w / 2;
    }


    //handle
    hipdnnHandle_t handle;
    CUDNN_CALL(hipdnnCreate(&handle));

    //input
    hipdnnTensorDescriptor_t input_descriptor;
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&input_descriptor));
    CUDNN_CALL(hipdnnSetTensor4dDescriptor(input_descriptor,
        dataformat,
        HIPDNN_DATA_FLOAT,
        input_n,
        input_c,
        input_h,
        input_w));


    //������
    hipdnnFilterDescriptor_t filter_descriptor;
    CUDNN_CALL(hipdnnCreateFilterDescriptor(&filter_descriptor));
    CUDNN_CALL(hipdnnSetFilter4dDescriptor(filter_descriptor,
        HIPDNN_DATA_FLOAT,
        dataformat,
        filter_n,
        filter_c,
        filter_h,
        filter_w));


    //��ķ�ʽ��������padding
    hipdnnConvolutionDescriptor_t conv_descriptor;
    CUDNN_CALL(hipdnnCreateConvolutionDescriptor(&conv_descriptor));
    CUDNN_CALL(hipdnnSetConvolution2dDescriptor(conv_descriptor,
        pad_h, pad_w, // zero-padding
        u, v, // stride
        1, 1,
        HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));

    //����output��4��ά��
    CUDNN_CALL(hipdnnGetConvolution2dForwardOutputDim(
        conv_descriptor,
        input_descriptor,
        filter_descriptor,
        &out_n,
        &out_c,
        &out_h,
        &out_w));

  

    if (dataformat == 0) {
        assert(doutput->shape[0] == out_n);
        assert(doutput->shape[1] == out_c);
        assert(1 == out_h);
        assert(doutput->shape[2] == out_w);
    }
    else {
        assert(doutput->shape[0] == out_n);
        assert(doutput->shape[2] == out_c);
        assert(1 == out_h);
        assert(doutput->shape[1] == out_w);
    }


    //output��Ϣ
    hipdnnTensorDescriptor_t output_descriptor;
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&output_descriptor));
    CUDNN_CALL(hipdnnSetTensor4dDescriptor(output_descriptor,
        dataformat,
        HIPDNN_DATA_FLOAT,
        out_n,
        out_c,
        out_h,
        out_w));

    //�������㷨
    hipdnnConvolutionBwdFilterAlgo_t  algo1;
    CUDNN_CALL(hipdnnGetConvolutionBackwardFilterAlgorithm(handle,
        input_descriptor,
        output_descriptor,
        conv_descriptor,
        filter_descriptor,
        HIPDNN_CONVOLUTION_BWD_FILTER_PREFER_FASTEST,
        0,
        &algo1));

    hipdnnConvolutionBwdDataAlgo_t algo2;
    CUDNN_CALL(hipdnnGetConvolutionBackwardDataAlgorithm(handle,
        filter_descriptor,
        output_descriptor,
        conv_descriptor,
        input_descriptor,
        HIPDNN_CONVOLUTION_BWD_DATA_PREFER_FASTEST,
        0,
        &algo2));

    //׼����������Ŀռ�


    size_t workspace_size1= 0;
    CUDNN_CALL(hipdnnGetConvolutionBackwardFilterWorkspaceSize(handle,
        input_descriptor,
        output_descriptor,
        conv_descriptor,
        filter_descriptor,
        algo1,
        &workspace_size1));
    void* workspace1= nullptr;
    hipMalloc(&workspace1, workspace_size1);

    size_t workspace_size2 = 0;
    CUDNN_CALL(hipdnnGetConvolutionBackwardDataWorkspaceSize(handle,
        filter_descriptor,
        output_descriptor,
        conv_descriptor,
        input_descriptor,
        algo2,
        &workspace_size2));
    void* workspace2 = nullptr;
    hipMalloc(&workspace2, workspace_size2);



    // convolution
    auto alpha = 1.0f, beta = 0.0f;
    CUDNN_CALL(hipdnnConvolutionBackwardFilter(handle,
        &alpha, //x*w����
        input_descriptor,
        input->data,
        output_descriptor,
        doutput->data,
        conv_descriptor,
        algo1,
        workspace1,
        workspace_size1,
        &beta, //y����,y�������ݽ������ţ�
        filter_descriptor,
        dfilter->data));



    CUDNN_CALL(hipdnnConvolutionBackwardData(handle,
        &alpha, //x*w����
        filter_descriptor,
        filter->data,
        output_descriptor,
        doutput->data,
        conv_descriptor,
        algo2,
        workspace2,
        workspace_size2,
        &beta, //y����,y�������ݽ������ţ�
        input_descriptor,
        dinput->data));


    //�ڴ�
    hipFree(workspace1);
    hipFree(workspace2);
    CUDNN_CALL(hipdnnDestroyTensorDescriptor(input_descriptor));
    CUDNN_CALL(hipdnnDestroyTensorDescriptor(output_descriptor));
    CUDNN_CALL(hipdnnDestroyConvolutionDescriptor(conv_descriptor));
    CUDNN_CALL(hipdnnDestroyFilterDescriptor(filter_descriptor));
    CUDNN_CALL(hipdnnDestroy(handle));

    return 0;

}





//4ά
int DLGpuConvolution2DForward(const DLArrayHandle input,
    const DLArrayHandle filter,
    DLArrayHandle output,
    hipdnnTensorFormat_t dataformat,
    const paddingStatus_t padding,
    const int u,          /* vertical filter stride */
    const int v          /* horizontal filter stride */){

    assert(input->ndim == 4);
    assert(filter->ndim == 4);


    int input_n = input->shape[0];
    int input_c = input->shape[3];
    int input_h = input->shape[1];
    int input_w = input->shape[2];

    int filter_n = filter->shape[0];
    int filter_c = filter->shape[3];
    int filter_h = filter->shape[1];
    int filter_w = filter->shape[2];


    if (dataformat == 0) {
        input_n = input->shape[0];
        input_c = input->shape[1];
        input_h = input->shape[2];
        input_w = input->shape[3];

        filter_n = filter->shape[0];
        filter_c = filter->shape[1];
        filter_h = filter->shape[2];
        filter_w = filter->shape[3];
    }



    int out_n;
    int out_c;
    int out_h;
    int out_w;

    int pad_h = 0;
    int pad_w = 0;

    if (padding == 1) {
        pad_h = filter_h / 2;
        pad_w = filter_w / 2;
    }


    //handle
    hipdnnHandle_t handle;
    CUDNN_CALL(hipdnnCreate(&handle));

    //input
    hipdnnTensorDescriptor_t input_descriptor;
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&input_descriptor));
    CUDNN_CALL(hipdnnSetTensor4dDescriptor(input_descriptor,
        dataformat,
        HIPDNN_DATA_FLOAT,
        input_n, 
        input_c, 
        input_h, 
        input_w));


    //������
    hipdnnFilterDescriptor_t filter_descriptor;
    CUDNN_CALL(hipdnnCreateFilterDescriptor(&filter_descriptor));
    CUDNN_CALL(hipdnnSetFilter4dDescriptor(filter_descriptor,
        HIPDNN_DATA_FLOAT,
        dataformat,
        filter_n, 
        filter_c, 
        filter_h, 
        filter_w));


    //��ķ�ʽ��������padding
    hipdnnConvolutionDescriptor_t conv_descriptor;
    CUDNN_CALL(hipdnnCreateConvolutionDescriptor(&conv_descriptor));
    CUDNN_CALL(hipdnnSetConvolution2dDescriptor(conv_descriptor,
        pad_h, pad_w, // zero-padding
        u, v, // stride
        1, 1,
        HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));

    //����output��4��ά��
    CUDNN_CALL(hipdnnGetConvolution2dForwardOutputDim(
        conv_descriptor,
        input_descriptor,
        filter_descriptor,
        &out_n,
        &out_c,
        &out_h,
        &out_w));

   

    if (dataformat == 0) {
        assert(output->shape[0] == out_n);
        assert(output->shape[1] == out_c);
        assert(output->shape[2] == out_h);
        assert(output->shape[3] == out_w);
    }
    else {
        assert(output->shape[0] == out_n);
        assert(output->shape[3] == out_c);
        assert(output->shape[1] == out_h);
        assert(output->shape[2] == out_w);
    }


    //output��Ϣ
    hipdnnTensorDescriptor_t output_descriptor;
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&output_descriptor));
    CUDNN_CALL(hipdnnSetTensor4dDescriptor(output_descriptor,
        dataformat,
        HIPDNN_DATA_FLOAT,
        out_n,
        out_c,
        out_h,
        out_w));

    //�������㷨
    hipdnnConvolutionFwdAlgo_t algo;
    CUDNN_CALL(hipdnnGetConvolutionForwardAlgorithm(handle,
        input_descriptor,
        filter_descriptor,
        conv_descriptor,
        output_descriptor,
        HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST,
        0,
        &algo));

    //׼����������Ŀռ�
    size_t workspace_size = 0;
    CUDNN_CALL(hipdnnGetConvolutionForwardWorkspaceSize(handle,
        input_descriptor,
        filter_descriptor,
        conv_descriptor,
        output_descriptor,
        algo,
        &workspace_size));
    void* workspace = nullptr;
    hipMalloc(&workspace, workspace_size);


    // convolution
    auto alpha = 1.0f, beta = 0.0f;
    CUDNN_CALL(hipdnnConvolutionForward(handle,
        &alpha, //x*w����
        input_descriptor,
        input->data,
        filter_descriptor,
        filter->data,
        conv_descriptor,
        algo,
        workspace,
        workspace_size,
        &beta, //y����,y�������ݽ������ţ�
        output_descriptor,
        output->data));

    //�ڴ�
    hipFree(workspace);
    CUDNN_CALL(hipdnnDestroyTensorDescriptor(input_descriptor));
    CUDNN_CALL(hipdnnDestroyTensorDescriptor(output_descriptor));
    CUDNN_CALL(hipdnnDestroyConvolutionDescriptor(conv_descriptor));
    CUDNN_CALL(hipdnnDestroyFilterDescriptor(filter_descriptor));
    CUDNN_CALL(hipdnnDestroy(handle));
    return 0;
}


int DLGpuConvolution2DBackward(const DLArrayHandle input,
    const DLArrayHandle doutput,
    const DLArrayHandle filter,
    DLArrayHandle dfilter,
    DLArrayHandle dinput,
    hipdnnTensorFormat_t dataformat,
    const paddingStatus_t padding,
    const int u,          /* vertical filter stride */
    const int v          /* horizontal filter stride */){

    assert(input->ndim == 4);
    assert(filter->ndim == 4);

    int input_n = input->shape[0];
    int input_c = input->shape[3];
    int input_h = input->shape[1];
    int input_w = input->shape[2];

    int filter_n = filter->shape[0];
    int filter_c = filter->shape[3];
    int filter_h = filter->shape[1];
    int filter_w = filter->shape[2];


    if (dataformat == 0) {
        input_n = input->shape[0];
        input_c = input->shape[1];
        input_h = input->shape[2];
        input_w = input->shape[3];

        filter_n = filter->shape[0];
        filter_c = filter->shape[1];
        filter_h = filter->shape[2];
        filter_w = filter->shape[3];
    }





    int out_n;
    int out_c;
    int out_h;
    int out_w;

    int pad_h = 0;
    int pad_w = 0;

    if (padding == 1) {
        pad_h = filter_h / 2;
        pad_w = filter_w / 2;
    }


    //handle
    hipdnnHandle_t handle;
    CUDNN_CALL(hipdnnCreate(&handle));

    //input
    hipdnnTensorDescriptor_t input_descriptor;
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&input_descriptor));
    CUDNN_CALL(hipdnnSetTensor4dDescriptor(input_descriptor,
        dataformat,
        HIPDNN_DATA_FLOAT,
        input_n,
        input_c,
        input_h,
        input_w));


    //������
    hipdnnFilterDescriptor_t filter_descriptor;
    CUDNN_CALL(hipdnnCreateFilterDescriptor(&filter_descriptor));
    CUDNN_CALL(hipdnnSetFilter4dDescriptor(filter_descriptor,
        HIPDNN_DATA_FLOAT,
        dataformat,
        filter_n,
        filter_c,
        filter_h,
        filter_w));


    //��ķ�ʽ��������padding
    hipdnnConvolutionDescriptor_t conv_descriptor;
    CUDNN_CALL(hipdnnCreateConvolutionDescriptor(&conv_descriptor));
    CUDNN_CALL(hipdnnSetConvolution2dDescriptor(conv_descriptor,
        pad_h, pad_w, // zero-padding
        u, v, // stride
        1, 1,
        HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));

    //����output��4��ά��
    CUDNN_CALL(hipdnnGetConvolution2dForwardOutputDim(
        conv_descriptor,
        input_descriptor,
        filter_descriptor,
        &out_n,
        &out_c,
        &out_h,
        &out_w));

    assert(doutput->shape[0] == out_n);
    assert(doutput->shape[1] == out_c);
    assert(doutput->shape[2] == out_h);
    assert(doutput->shape[3] == out_w);




    //output��Ϣ
    hipdnnTensorDescriptor_t output_descriptor;
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&output_descriptor));
    CUDNN_CALL(hipdnnSetTensor4dDescriptor(output_descriptor,
        dataformat,
        HIPDNN_DATA_FLOAT,
        out_n,
        out_c,
        out_h,
        out_w));

    hipdnnConvolutionBwdFilterAlgo_t  algo1;
    CUDNN_CALL(hipdnnGetConvolutionBackwardFilterAlgorithm(handle,
        input_descriptor,
        output_descriptor,
        conv_descriptor,
        filter_descriptor,
        HIPDNN_CONVOLUTION_BWD_FILTER_PREFER_FASTEST,
        0,
        &algo1));

    hipdnnConvolutionBwdDataAlgo_t algo2;
    CUDNN_CALL(hipdnnGetConvolutionBackwardDataAlgorithm(handle,
        filter_descriptor,
        output_descriptor,
        conv_descriptor,
        input_descriptor,
        HIPDNN_CONVOLUTION_BWD_DATA_PREFER_FASTEST,
        0,
        &algo2));

    //׼����������Ŀռ�


    size_t workspace_size1= 0;
    CUDNN_CALL(hipdnnGetConvolutionBackwardFilterWorkspaceSize(handle,
        input_descriptor,
        output_descriptor,
        conv_descriptor,
        filter_descriptor,
        algo1,
        &workspace_size1));
    void* workspace1= nullptr;
    hipMalloc(&workspace1, workspace_size1);

    size_t workspace_size2 = 0;
    CUDNN_CALL(hipdnnGetConvolutionBackwardDataWorkspaceSize(handle,
        filter_descriptor,
        output_descriptor,
        conv_descriptor,
        input_descriptor,
        algo2,
        &workspace_size2));
    void* workspace2 = nullptr;
    hipMalloc(&workspace2, workspace_size2);



    // convolution
    auto alpha = 1.0f, beta = 0.0f;
    CUDNN_CALL(hipdnnConvolutionBackwardFilter(handle,
        &alpha, //x*w����
        input_descriptor,
        input->data,
        output_descriptor,
        doutput->data,
        conv_descriptor,
        algo1,
        workspace1,
        workspace_size1,
        &beta, //y����,y�������ݽ������ţ�
        filter_descriptor,
        dfilter->data));



    CUDNN_CALL(hipdnnConvolutionBackwardData(handle,
        &alpha, //x*w����
        filter_descriptor,
        filter->data,
        output_descriptor,
        doutput->data,
        conv_descriptor,
        algo2,
        workspace2,
        workspace_size2,
        &beta, //y����,y�������ݽ������ţ�
        input_descriptor,
        dinput->data));


    //�ڴ�
    hipFree(workspace1);
    hipFree(workspace2);
    CUDNN_CALL(hipdnnDestroyTensorDescriptor(input_descriptor));
    CUDNN_CALL(hipdnnDestroyTensorDescriptor(output_descriptor));
    CUDNN_CALL(hipdnnDestroyConvolutionDescriptor(conv_descriptor));
    CUDNN_CALL(hipdnnDestroyFilterDescriptor(filter_descriptor));
    CUDNN_CALL(hipdnnDestroy(handle));
    return 0;
}

//5ά
int DLGpuConvolution3DForward(const DLArrayHandle input,
    const DLArrayHandle filter,
    DLArrayHandle output,
    hipdnnTensorFormat_t dataformat,
    const paddingStatus_t padding,
    const int s1,          
    const int s2,     
    const int s3) {

    assert(input->ndim == 5);
    assert(filter->ndim == 5);

    int* input_shape, * output_shape, * filter_shape, * inputstrides,*outputstrides;

    int* padA, * filterStrideA, * dilationA;

    input_shape = (int*)malloc(sizeof(int) * 5);
    filter_shape = (int*)malloc(sizeof(int) * 5);
    output_shape = (int*)malloc(sizeof(int) * 5);
    inputstrides = (int*)malloc(sizeof(int) * 5);
    outputstrides = (int*)malloc(sizeof(int) * 5);
    padA = (int*)malloc(sizeof(int) * 3);
    filterStrideA = (int*)malloc(sizeof(int) * 3);
    dilationA = (int*)malloc(sizeof(int) * 3);

    for (int i=0;i<5;i++)
    {
        input_shape[i]=input->shape[i];
        filter_shape[i]=filter->shape[i];
    }


    for (int i = 0; i < 3; i++) {
        padA[i] = 0;
        dilationA[i] = 1;
    }

    if (padding == 1) {
        for (int i = 0; i < 3; i++) {
            padA[i] = filter_shape[i+2]/2;
            
        }
    }


    filterStrideA[0] = s1;
    filterStrideA[1] = s2;
    filterStrideA[2] = s3;


    inputstrides[4] = 1;
    for (int i = 1; i < 5; i++) {
        inputstrides[4 - i] = inputstrides[5 - i] * input_shape[5 - i];
    }



     hipdnnHandle_t handle;
    CUDNN_CALL(hipdnnCreate(&handle));


    //input
    hipdnnTensorDescriptor_t input_descriptor;
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&input_descriptor));
    CUDNN_CALL(cudnnSetTensorNdDescriptorEx(input_descriptor,
        dataformat,
        HIPDNN_DATA_FLOAT,
        5,
        input_shape));

    //�˺���
    hipdnnFilterDescriptor_t filter_descriptor;
    CUDNN_CALL(hipdnnCreateFilterDescriptor(&filter_descriptor));
    CUDNN_CALL(hipdnnSetFilterNdDescriptor(filter_descriptor,
        HIPDNN_DATA_FLOAT,
        dataformat,
        5,
        filter_shape));

    //��ķ�ʽ��������padding
    hipdnnConvolutionDescriptor_t conv_descriptor;
    CUDNN_CALL(hipdnnCreateConvolutionDescriptor(&conv_descriptor));
    CUDNN_CALL(hipdnnSetConvolutionNdDescriptor(conv_descriptor,
        3,
        padA,
        filterStrideA,
        dilationA,
        HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));

    //output��shape
    CUDNN_CALL(cudnnGetConvolutionNdForwardOutputDim(
        conv_descriptor,
        input_descriptor,
        filter_descriptor,
        5,
        output_shape));


    assert(output->shape[0] == output_shape[0]);
    assert(output->shape[1] == output_shape[1]);
    assert(output->shape[2] == output_shape[2]);
    assert(output->shape[3] == output_shape[3]);
    assert(output->shape[4] == output_shape[4]);


    outputstrides[4] = 1;
    for (int i = 1; i < 5; i++) {
        outputstrides[4 - i] = outputstrides[5 - i] * output_shape[5 - i];
    }


    //output
    hipdnnTensorDescriptor_t output_descriptor;
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&output_descriptor));
    CUDNN_CALL(cudnnSetTensorNdDescriptorEx(output_descriptor,
        dataformat,
        HIPDNN_DATA_FLOAT,
        5,
        output_shape));

    //�������㷨
    hipdnnConvolutionFwdAlgo_t algo;
    CUDNN_CALL(hipdnnGetConvolutionForwardAlgorithm(handle,
        input_descriptor,
        filter_descriptor,
        conv_descriptor,
        output_descriptor,
        HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST,
        0,
        &algo));

    //׼����������Ŀռ�
    size_t workspace_size = 0;
    CUDNN_CALL(hipdnnGetConvolutionForwardWorkspaceSize(handle,
        input_descriptor,
        filter_descriptor,
        conv_descriptor,
        output_descriptor,
        algo,
        &workspace_size));
    void* workspace = nullptr;
    hipMalloc(&workspace, workspace_size);


    // convolution
    auto alpha = 1.0f, beta = 0.0f;
    CUDNN_CALL(hipdnnConvolutionForward(handle,
        &alpha, //x*w����
        input_descriptor,
        input->data,
        filter_descriptor,
        filter->data,
        conv_descriptor,
        algo,
        workspace,
        workspace_size,
        &beta, //y����,y�������ݽ������ţ�
        output_descriptor,
        output->data));

    //�ڴ�
    hipFree(workspace);
    CUDNN_CALL(hipdnnDestroyTensorDescriptor(input_descriptor));
    CUDNN_CALL(hipdnnDestroyTensorDescriptor(output_descriptor));
    CUDNN_CALL(hipdnnDestroyConvolutionDescriptor(conv_descriptor));
    CUDNN_CALL(hipdnnDestroyFilterDescriptor(filter_descriptor));
    CUDNN_CALL(hipdnnDestroy(handle));


   return 0;




}

int DLGpuConvolution3DBackward(const DLArrayHandle input,
    const DLArrayHandle doutput,
    const DLArrayHandle filter,
    DLArrayHandle dfilter,
    DLArrayHandle dinput,
    hipdnnTensorFormat_t dataformat,
    const paddingStatus_t padding,
    const int s1,
    const int s2,
    const int s3) {

    assert(input->ndim == 5);
    assert(filter->ndim == 5);

    int* input_shape, * output_shape, * filter_shape, * inputstrides,*outputstrides;

    int* padA, * filterStrideA, * dilationA;

    input_shape = (int*)malloc(sizeof(int) * 5);
    filter_shape = (int*)malloc(sizeof(int) * 5);
    output_shape = (int*)malloc(sizeof(int) * 5);
    inputstrides = (int*)malloc(sizeof(int) * 5);
    outputstrides = (int*)malloc(sizeof(int) * 5);
    padA = (int*)malloc(sizeof(int) * 3);
    filterStrideA = (int*)malloc(sizeof(int) * 3);
    dilationA = (int*)malloc(sizeof(int) * 3);

    for (int i=0;i<5;i++)
    {
        input_shape[i]=input->shape[i];
        filter_shape[i]=filter->shape[i];
    }


    for (int i = 0; i < 3; i++) {
        padA[i] = 0;
        dilationA[i] = 1;
    }

    if (padding == 1) {
        for (int i = 0; i < 3; i++) {
            padA[i] = filter_shape[i+2]/2;

        }
    }


    filterStrideA[0] = s1;
    filterStrideA[1] = s2;
    filterStrideA[2] = s3;


    inputstrides[4] = 1;
    for (int i = 1; i < 5; i++) {
        inputstrides[4 - i] = inputstrides[5 - i] * input_shape[5 - i];
    }



     hipdnnHandle_t handle;
    CUDNN_CALL(hipdnnCreate(&handle));


    //input
    hipdnnTensorDescriptor_t input_descriptor;
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&input_descriptor));
    CUDNN_CALL(cudnnSetTensorNdDescriptorEx(input_descriptor,
        dataformat,
        HIPDNN_DATA_FLOAT,
        5,
        input_shape));

    //�˺���
    hipdnnFilterDescriptor_t filter_descriptor;
    CUDNN_CALL(hipdnnCreateFilterDescriptor(&filter_descriptor));
    CUDNN_CALL(hipdnnSetFilterNdDescriptor(filter_descriptor,
        HIPDNN_DATA_FLOAT,
        dataformat,
        5,
        filter_shape));

    //��ķ�ʽ��������padding
    hipdnnConvolutionDescriptor_t conv_descriptor;
    CUDNN_CALL(hipdnnCreateConvolutionDescriptor(&conv_descriptor));
    CUDNN_CALL(hipdnnSetConvolutionNdDescriptor(conv_descriptor,
        3,
        padA,
        filterStrideA,
        dilationA,
        HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));

    //output��shape
    CUDNN_CALL(cudnnGetConvolutionNdForwardOutputDim(
        conv_descriptor,
        input_descriptor,
        filter_descriptor,
        5,
        output_shape));


    assert(doutput->shape[0] == output_shape[0]);
    assert(doutput->shape[1] == output_shape[1]);
    assert(doutput->shape[2] == output_shape[2]);
    assert(doutput->shape[3] == output_shape[3]);
    assert(doutput->shape[4] == output_shape[4]);


    outputstrides[4] = 1;
    for (int i = 1; i < 5; i++) {
        outputstrides[4 - i] = outputstrides[5 - i] * output_shape[5 - i];
    }


    //output
    hipdnnTensorDescriptor_t output_descriptor;
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&output_descriptor));
    CUDNN_CALL(cudnnSetTensorNdDescriptorEx(output_descriptor,
        dataformat,
        HIPDNN_DATA_FLOAT,
        5,
        output_shape));

       hipdnnConvolutionBwdFilterAlgo_t  algo1;
    CUDNN_CALL(hipdnnGetConvolutionBackwardFilterAlgorithm(handle,
        input_descriptor,
        output_descriptor,
        conv_descriptor,
        filter_descriptor,
        HIPDNN_CONVOLUTION_BWD_FILTER_PREFER_FASTEST,
        0,
        &algo1));

    hipdnnConvolutionBwdDataAlgo_t algo2;
    CUDNN_CALL(hipdnnGetConvolutionBackwardDataAlgorithm(handle,
        filter_descriptor,
        output_descriptor,
        conv_descriptor,
        input_descriptor,
        HIPDNN_CONVOLUTION_BWD_DATA_PREFER_FASTEST,
        0,
        &algo2));

    //׼����������Ŀռ�


    size_t workspace_size1= 0;
    CUDNN_CALL(hipdnnGetConvolutionBackwardFilterWorkspaceSize(handle,
        input_descriptor,
        output_descriptor,
        conv_descriptor,
        filter_descriptor,
        algo1,
        &workspace_size1));
    void* workspace1= nullptr;
    hipMalloc(&workspace1, workspace_size1);

    size_t workspace_size2 = 0;
    CUDNN_CALL(hipdnnGetConvolutionBackwardDataWorkspaceSize(handle,
        filter_descriptor,
        output_descriptor,
        conv_descriptor,
        input_descriptor,
        algo2,
        &workspace_size2));
    void* workspace2 = nullptr;
    hipMalloc(&workspace2, workspace_size2);



    // convolution
    auto alpha = 1.0f, beta = 0.0f;
    CUDNN_CALL(hipdnnConvolutionBackwardFilter(handle,
        &alpha, //x*w����
        input_descriptor,
        input->data,
        output_descriptor,
        doutput->data,
        conv_descriptor,
        algo1,
        workspace1,
        workspace_size1,
        &beta, //y����,y�������ݽ������ţ�
        filter_descriptor,
        dfilter->data));



    CUDNN_CALL(hipdnnConvolutionBackwardData(handle,
        &alpha, //x*w����
        filter_descriptor,
        filter->data,
        output_descriptor,
        doutput->data,
        conv_descriptor,
        algo2,
        workspace2,
        workspace_size2,
        &beta, //y����,y�������ݽ������ţ�
        input_descriptor,
        dinput->data));


    //�ڴ�
    hipFree(workspace1);
    hipFree(workspace2);
    CUDNN_CALL(hipdnnDestroyTensorDescriptor(input_descriptor));
    CUDNN_CALL(hipdnnDestroyTensorDescriptor(output_descriptor));
    CUDNN_CALL(hipdnnDestroyConvolutionDescriptor(conv_descriptor));
    CUDNN_CALL(hipdnnDestroyFilterDescriptor(filter_descriptor));
    CUDNN_CALL(hipdnnDestroy(handle));
    return 0;
}



int DLGpuPooling1DForward(const DLArrayHandle input,
    DLArrayHandle output,
    hipdnnTensorFormat_t dataformat,
    const int padding_w,
    const int v,
    const int filter_w) {

    int padding_h = 0;
    int u = 1;
    int filter_h = 1;


    int input_n = input->shape[0];
    int input_c = input->shape[2];
    int input_h = 1;
    int input_w = input->shape[1];



    if (dataformat == 0) {
        input_n = input->shape[0];
        input_c = input->shape[1];
        input_h = 1;
        input_w = input->shape[2];
    }




    int output_n;
    int output_c;
    int output_h;
    int output_w;


    //handle
    hipdnnHandle_t handle;
    CUDNN_CALL(hipdnnCreate(&handle));


    hipdnnPoolingDescriptor_t pool_descriptor;
    CUDNN_CALL(hipdnnCreatePoolingDescriptor(&pool_descriptor));
    CUDNN_CALL(hipdnnSetPooling2dDescriptor(pool_descriptor,
        HIPDNN_POOLING_MAX,
        HIPDNN_NOT_PROPAGATE_NAN,
        filter_h, filter_w,
        padding_h, padding_w, // zero-padding
        u, v // stride
    ));


    //input
    hipdnnTensorDescriptor_t input_descriptor;
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&input_descriptor));
    CUDNN_CALL(hipdnnSetTensor4dDescriptor(input_descriptor,
        HIPDNN_TENSOR_NHWC,
        HIPDNN_DATA_FLOAT,
        input_n,
        input_c,
        input_h,
        input_w));



    CUDNN_CALL(hipdnnGetPooling2dForwardOutputDim(
        pool_descriptor,
        input_descriptor,
        &output_n,
        &output_c,
        &output_h,
        &output_w));

    if (dataformat == 0) {
        assert(output->shape[0] == output_n);
        assert(output->shape[1] == output_c);
        assert(1 == output_h);
        assert(output->shape[2] == output_w);
    }
    else {
        assert(output->shape[0] == output_n);
        assert(output->shape[2] == output_c);
        assert(1 == output_h);
        assert(output->shape[1] == output_w);
    }
    

    hipdnnTensorDescriptor_t output_descriptor;
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&output_descriptor));
    CUDNN_CALL(hipdnnSetTensor4dDescriptor(output_descriptor,
        HIPDNN_TENSOR_NHWC,
        HIPDNN_DATA_FLOAT,
        output_n,
        output_c,
        output_h,
        output_w));


    auto alpha = 1.0f, beta = 1.0f;
    CUDNN_CALL(hipdnnPoolingForward(handle,
        pool_descriptor,
        &alpha,
        input_descriptor,
        input->data,
        &beta,
        output_descriptor,
        output->data));

    CUDNN_CALL(hipdnnDestroyTensorDescriptor(input_descriptor));
    CUDNN_CALL(hipdnnDestroyTensorDescriptor(output_descriptor));
    CUDNN_CALL(hipdnnDestroyPoolingDescriptor(pool_descriptor));
    CUDNN_CALL(hipdnnDestroy(handle));
    return 0;
}

int DLGpuPooling1DBackward(const DLArrayHandle input,
    const DLArrayHandle output,
    const DLArrayHandle doutput,
    DLArrayHandle dinput,
    hipdnnTensorFormat_t dataformat,
    const int padding_w,
    const int v,
    const int filter_w) {

    int padding_h = 0;
    int u = 1;
    int filter_h = 1;


    int input_n = input->shape[0];
    int input_c = input->shape[2];
    int input_h = 1;
    int input_w = input->shape[1];

    if (dataformat == 0) {
        input_n = input->shape[0];
        input_c = input->shape[1];
        input_h = 1;
        input_w = input->shape[2];
    }


    int output_n;
    int output_c;
    int output_h;
    int output_w;


    //handle
    hipdnnHandle_t handle;
    CUDNN_CALL(hipdnnCreate(&handle));


    hipdnnPoolingDescriptor_t pool_descriptor;
    CUDNN_CALL(hipdnnCreatePoolingDescriptor(&pool_descriptor));
    CUDNN_CALL(hipdnnSetPooling2dDescriptor(pool_descriptor,
        HIPDNN_POOLING_MAX,
        HIPDNN_NOT_PROPAGATE_NAN,
        filter_h, filter_w,
        padding_h, padding_w, // zero-padding
        u, v // stride
    ));


    //input
    hipdnnTensorDescriptor_t input_descriptor;
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&input_descriptor));
    CUDNN_CALL(hipdnnSetTensor4dDescriptor(input_descriptor,
        dataformat,
        HIPDNN_DATA_FLOAT,
        input_n,
        input_c,
        input_h,
        input_w));



    CUDNN_CALL(hipdnnGetPooling2dForwardOutputDim(
        pool_descriptor,
        input_descriptor,
        &output_n,
        &output_c,
        &output_h,
        &output_w));


   

    if (dataformat == 0) {
        assert(output->shape[0] == output_n);
        assert(output->shape[1] == output_c);
        assert(1 == output_h);
        assert(output->shape[2] == output_w);
    }
    else {
        assert(output->shape[0] == output_n);
        assert(output->shape[2] == output_c);
        assert(1 == output_h);
        assert(output->shape[1] == output_w);
    }

    hipdnnTensorDescriptor_t output_descriptor;
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&output_descriptor));
    CUDNN_CALL(hipdnnSetTensor4dDescriptor(output_descriptor,
        dataformat,
        HIPDNN_DATA_FLOAT,
        output_n,
        output_c,
        output_h,
        output_w));


    auto alpha = 1.0f, beta = 1.0f;
    CUDNN_CALL(hipdnnPoolingBackward(handle,
        pool_descriptor,
        &alpha,
        output_descriptor,
        output->data,
        output_descriptor,
        doutput->data,
        input_descriptor,
        input->data,
        &beta,
        input_descriptor,
        dinput->data));

    CUDNN_CALL(hipdnnDestroyTensorDescriptor(input_descriptor));
    CUDNN_CALL(hipdnnDestroyTensorDescriptor(output_descriptor));
    CUDNN_CALL(hipdnnDestroyPoolingDescriptor(pool_descriptor));
    CUDNN_CALL(hipdnnDestroy(handle));
    return 0;
}



int DLGpuPooling2DForward(const DLArrayHandle input,
    DLArrayHandle output,
    hipdnnTensorFormat_t dataformat,
    const int padding_h,
    const int padding_w,
    const int u,
    const int v,
    const int filter_h,
    const int filter_w)
{

    int input_n = input->shape[0];
    int input_c = input->shape[3];
    int input_h = input->shape[1];
    int input_w = input->shape[2];
    if (dataformat == 0) {
        input_n = input->shape[0];
        input_c = input->shape[1];
        input_h = input->shape[2];
        input_w = input->shape[3];
    }



    int output_n;
    int output_c;
    int output_h;
    int output_w;

    //handle
    hipdnnHandle_t handle;
    CUDNN_CALL(hipdnnCreate(&handle));


    hipdnnPoolingDescriptor_t pool_descriptor;
    CUDNN_CALL(hipdnnCreatePoolingDescriptor(&pool_descriptor));
    CUDNN_CALL(hipdnnSetPooling2dDescriptor(pool_descriptor,
        HIPDNN_POOLING_MAX,
        HIPDNN_NOT_PROPAGATE_NAN,
        filter_h, filter_w,
        padding_h, padding_w, // zero-padding
        u, v // stride
    ));
    //input
    hipdnnTensorDescriptor_t input_descriptor;
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&input_descriptor));
    CUDNN_CALL(hipdnnSetTensor4dDescriptor(input_descriptor,
        dataformat,
        HIPDNN_DATA_FLOAT,
        input_n,
        input_c,
        input_h,
        input_w));




    CUDNN_CALL(hipdnnGetPooling2dForwardOutputDim(
        pool_descriptor,
        input_descriptor,
        &output_n,
        &output_c,
        &output_h,
        &output_w));

    hipdnnTensorDescriptor_t output_descriptor;
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&output_descriptor));
    CUDNN_CALL(hipdnnSetTensor4dDescriptor(output_descriptor,
        dataformat,
        HIPDNN_DATA_FLOAT,
        output_n,
        output_c,
        output_h,
        output_w));


    auto alpha = 1.0f, beta = 1.0f;
    CUDNN_CALL(hipdnnPoolingForward(handle,
        pool_descriptor,
        &alpha,
        input_descriptor,
        input->data,
        &beta,
        output_descriptor,
        output->data));

    CUDNN_CALL(hipdnnDestroyTensorDescriptor(input_descriptor));
    CUDNN_CALL(hipdnnDestroyTensorDescriptor(output_descriptor));
    CUDNN_CALL(hipdnnDestroyPoolingDescriptor(pool_descriptor));
    CUDNN_CALL(hipdnnDestroy(handle));
   return 0;
}



int DLGpuPooling2DBackward(const DLArrayHandle input,
    const DLArrayHandle output,
    const DLArrayHandle doutput,
    DLArrayHandle dinput,
    hipdnnTensorFormat_t dataformat,
    const int padding_h,
    const int padding_w,
    const int u,
    const int v,
    const int filter_h,
    const int filter_w)
{
    int input_n = input->shape[0];
    int input_c = input->shape[3];
    int input_h = input->shape[1];
    int input_w = input->shape[2];

    if (dataformat == 0) {
        input_n = input->shape[0];
        input_c = input->shape[1];
        input_h = input->shape[2];
        input_w = input->shape[3];
    }



    int output_n;
    int output_c;
    int output_h;
    int output_w;

    //handle
    hipdnnHandle_t handle;
    CUDNN_CALL(hipdnnCreate(&handle));


    hipdnnPoolingDescriptor_t pool_descriptor;
    CUDNN_CALL(hipdnnCreatePoolingDescriptor(&pool_descriptor));
    CUDNN_CALL(hipdnnSetPooling2dDescriptor(pool_descriptor,
        HIPDNN_POOLING_MAX,
        HIPDNN_NOT_PROPAGATE_NAN,
        filter_h, filter_w,
        padding_h, padding_w, // zero-padding
        u, v // stride
    ));
    //input
    hipdnnTensorDescriptor_t input_descriptor;
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&input_descriptor));
    CUDNN_CALL(hipdnnSetTensor4dDescriptor(input_descriptor,
        dataformat,
        HIPDNN_DATA_FLOAT,
        input_n,
        input_c,
        input_h,
        input_w));




    CUDNN_CALL(hipdnnGetPooling2dForwardOutputDim(
        pool_descriptor,
        input_descriptor,
        &output_n,
        &output_c,
        &output_h,
        &output_w));

    hipdnnTensorDescriptor_t output_descriptor;
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&output_descriptor));
    CUDNN_CALL(hipdnnSetTensor4dDescriptor(output_descriptor,
        dataformat,
        HIPDNN_DATA_FLOAT,
        output_n,
        output_c,
        output_h,
        output_w));


    auto alpha = 1.0f, beta = 1.0f;
    CUDNN_CALL(hipdnnPoolingBackward(handle,
        pool_descriptor,
        &alpha,
        output_descriptor,
        output->data,
        output_descriptor,
        doutput->data,
        input_descriptor,
        input->data,
        &beta,
        input_descriptor,
        dinput->data));

    CUDNN_CALL(hipdnnDestroyTensorDescriptor(input_descriptor));
    CUDNN_CALL(hipdnnDestroyTensorDescriptor(output_descriptor));
    CUDNN_CALL(hipdnnDestroyPoolingDescriptor(pool_descriptor));
    CUDNN_CALL(hipdnnDestroy(handle));
   return 0;
}




int DLGpuPooling3DForward(const DLArrayHandle input,
    DLArrayHandle output,
    hipdnnTensorFormat_t dataformat,
    const int padding1,
    const int padding2,
    const int padding3,
    const int s1,
    const int s2,
    const int s3,
    const int filter1,
    const int filter2,
    const int filter3)
{
    assert(input->ndim == 5);


    int* input_shape, * output_shape, * filter_shape, * inputstrides, * outputstrides;


    int* padA, * filterStrideA;
    input_shape = (int*)malloc(sizeof(int) * 5);
    filter_shape = (int*)malloc(sizeof(int) * 3);
    output_shape = (int*)malloc(sizeof(int) * 5);
    inputstrides = (int*)malloc(sizeof(int) * 5);
    outputstrides = (int*)malloc(sizeof(int) * 5);
    padA = (int*)malloc(sizeof(int) * 3);
    filterStrideA = (int*)malloc(sizeof(int) * 3);
    for(int i=0;i<5;i++)
    {
        input_shape[i]= input->shape[i];
    }
    filter_shape[0] = filter1;
    filter_shape[1] = filter2;
    filter_shape[2] = filter3;
    filterStrideA[0] = s1;
    filterStrideA[1] = s2;
    filterStrideA[2] = s3;
    padA[0] = padding1;
    padA[1] = padding2;
    padA[2] = padding3;


    inputstrides[4] = 1;
    for (int i = 1; i < 5; i++) {
        inputstrides[4 - i] = inputstrides[5 - i] * input_shape[5 - i];
    }

    //handle
    hipdnnHandle_t handle;
    CUDNN_CALL(hipdnnCreate(&handle));


    hipdnnPoolingDescriptor_t pool_descriptor;
    CUDNN_CALL(hipdnnCreatePoolingDescriptor(&pool_descriptor));
    CUDNN_CALL(hipdnnSetPoolingNdDescriptor(pool_descriptor,
        HIPDNN_POOLING_MAX,
        HIPDNN_NOT_PROPAGATE_NAN,
        3,
        filter_shape,
        padA,
        filterStrideA));
    

    //input
    hipdnnTensorDescriptor_t input_descriptor;
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&input_descriptor));
    CUDNN_CALL(cudnnSetTensorNdDescriptorEx(input_descriptor,
        dataformat,
        HIPDNN_DATA_FLOAT,
        5,
        input_shape));


    CUDNN_CALL(cudnnGetPoolingNdForwardOutputDim(
        pool_descriptor,
        input_descriptor,
        5,
        output_shape));

    assert(output->shape[0] == output_shape[0]);
    assert(output->shape[1] == output_shape[1]);
    assert(output->shape[2] == output_shape[2]);
    assert(output->shape[3] == output_shape[3]);
    assert(output->shape[4] == output_shape[4]);

    outputstrides[4] = 1;
    for (int i = 1; i < 5; i++) {
        outputstrides[4 - i] = outputstrides[5 - i] * output_shape[5 - i];
    }


    //output
    hipdnnTensorDescriptor_t output_descriptor;
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&output_descriptor));
    CUDNN_CALL(cudnnSetTensorNdDescriptorEx(output_descriptor,
        dataformat,
        HIPDNN_DATA_FLOAT,
        5,
        output_shape));


    auto alpha = 1.0f, beta = 1.0f;
    CUDNN_CALL(hipdnnPoolingForward(handle,
        pool_descriptor,
        &alpha,
        input_descriptor,
        input->data,
        &beta,
        output_descriptor,
        output->data));

    CUDNN_CALL(hipdnnDestroyTensorDescriptor(input_descriptor));
    CUDNN_CALL(hipdnnDestroyTensorDescriptor(output_descriptor));
    CUDNN_CALL(hipdnnDestroyPoolingDescriptor(pool_descriptor));
    CUDNN_CALL(hipdnnDestroy(handle));
    return 0;

}




int DLGpuPooling3DBackward(const DLArrayHandle input,
    const DLArrayHandle output,
    const DLArrayHandle doutput,
    DLArrayHandle dinput,
    hipdnnTensorFormat_t dataformat,
    const int padding1,
    const int padding2,
    const int padding3,
    const int s1,
    const int s2,
    const int s3,
    const int filter1,
    const int filter2,
    const int filter3)
{
    assert(input->ndim == 5);


    int* input_shape, * output_shape, * filter_shape, * inputstrides, * outputstrides;


    int* padA, * filterStrideA;
    input_shape = (int*)malloc(sizeof(int) * 5);
    filter_shape = (int*)malloc(sizeof(int) * 3);
    output_shape = (int*)malloc(sizeof(int) * 5);
    inputstrides = (int*)malloc(sizeof(int) * 5);
    outputstrides = (int*)malloc(sizeof(int) * 5);
    padA = (int*)malloc(sizeof(int) * 3);
    filterStrideA = (int*)malloc(sizeof(int) * 3);
    for(int i=0;i<5;i++)
    {
        input_shape[i]= input->shape[i];
    }
    filter_shape[0] = filter1;
    filter_shape[1] = filter2;
    filter_shape[2] = filter3;
    filterStrideA[0] = s1;
    filterStrideA[1] = s2;
    filterStrideA[2] = s3;
    padA[0] = padding1;
    padA[1] = padding2;
    padA[2] = padding3;


    inputstrides[4] = 1;
    for (int i = 1; i < 5; i++) {
        inputstrides[4 - i] = inputstrides[5 - i] * input_shape[5 - i];
    }

    //handle
    hipdnnHandle_t handle;
    CUDNN_CALL(hipdnnCreate(&handle));


    hipdnnPoolingDescriptor_t pool_descriptor;
    CUDNN_CALL(hipdnnCreatePoolingDescriptor(&pool_descriptor));
    CUDNN_CALL(hipdnnSetPoolingNdDescriptor(pool_descriptor,
        HIPDNN_POOLING_MAX,
        HIPDNN_NOT_PROPAGATE_NAN,
        3,
        filter_shape,
        padA,
        filterStrideA));


    //input
    hipdnnTensorDescriptor_t input_descriptor;
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&input_descriptor));
    CUDNN_CALL(cudnnSetTensorNdDescriptorEx(input_descriptor,
        dataformat,
        HIPDNN_DATA_FLOAT,
        5,
        input_shape));


    CUDNN_CALL(cudnnGetPoolingNdForwardOutputDim(
        pool_descriptor,
        input_descriptor,
        5,
        output_shape));

    assert(output->shape[0] == output_shape[0]);
    assert(output->shape[1] == output_shape[1]);
    assert(output->shape[2] == output_shape[2]);
    assert(output->shape[3] == output_shape[3]);
    assert(output->shape[4] == output_shape[4]);

    outputstrides[4] = 1;
    for (int i = 1; i < 5; i++) {
        outputstrides[4 - i] = outputstrides[5 - i] * output_shape[5 - i];
    }


    //output
    hipdnnTensorDescriptor_t output_descriptor;
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&output_descriptor));
    CUDNN_CALL(cudnnSetTensorNdDescriptorEx(output_descriptor,
        dataformat,
        HIPDNN_DATA_FLOAT,
        5,
        output_shape));


    auto alpha = 1.0f, beta = 1.0f;
    CUDNN_CALL(hipdnnPoolingBackward(handle,
        pool_descriptor,
        &alpha,
        output_descriptor,
        output->data,
        output_descriptor,
        doutput->data,
        input_descriptor,
        input->data,
        &beta,
        input_descriptor,
        dinput->data));

    CUDNN_CALL(hipdnnDestroyTensorDescriptor(input_descriptor));
    CUDNN_CALL(hipdnnDestroyTensorDescriptor(output_descriptor));
    CUDNN_CALL(hipdnnDestroyPoolingDescriptor(pool_descriptor));
    CUDNN_CALL(hipdnnDestroy(handle));
    return 0;

}







